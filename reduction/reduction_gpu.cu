#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */ 

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */
template <class T>
__global__ void
reduce0(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/* This version uses contiguous threads, but its interleaved
   addressing results in many shared memory bank conflicts.
*/
template <class T>
__global__ void
reduce1(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        int index = 2 * s * tid;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
template <class T>
__global__ void
reduce2(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
*/
template <class T>
__global__ void
reduce3(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockDim.x < n)
        mySum += g_idata[i+blockDim.x];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

/*
    This version uses the warp shuffle operation if available to reduce 
    warp synchronization. When shuffle is not available the final warp's
    worth of work is unrolled to reduce looping overhead.

    See http://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
    for additional information about using shuffle to perform a reduction
    within a warp.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize>
__global__ void
reduce4(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockSize < n)
        mySum += g_idata[i+blockSize];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }

    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

/*
    This version is completely unrolled, unless warp shuffle is available, then
    shuffle is used within a loop.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time. When shuffle is available, it is used to reduce warp synchronization.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize>
__global__ void
reduce5(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockSize < n)
        mySum += g_idata[i+blockSize];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks,
       int whichKernel, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    // choose which of the optimized versions of reduction to launch
    switch (whichKernel)
    {
        case 0:
            reduce0<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 1:
            reduce1<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 2:
            reduce2<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 3:
            reduce3<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 4:
            switch (threads)
            {
                case 512:
                    reduce4<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 256:
                    reduce4<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 128:
                    reduce4<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 64:
                    reduce4<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 32:
                    reduce4<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 16:
                    reduce4<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  8:
                    reduce4<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  4:
                    reduce4<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  2:
                    reduce4<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  1:
                    reduce4<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;
            }

            break;

        case 5:
        default:
            switch (threads)
            {
                case 512:
                    reduce5<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 256:
                    reduce5<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 128:
                    reduce5<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 64:
                    reduce5<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 32:
                    reduce5<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 16:
                    reduce5<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  8:
                    reduce5<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  4:
                    reduce5<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  2:
                    reduce5<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  1:
                    reduce5<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;
            }

            break;
    }
}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{

    //get device capability, to avoid block/grid size exceed the upper bound
    hipDeviceProp_t prop;
    int device;
    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&prop, device));

    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
        blocks = (n + threads - 1) / threads;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }

    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    if (whichKernel == 6)
    {
        blocks = MIN(maxBlocks, blocks);
    }
}

int main(int argc, char *argv[])
{
    int n = 1 << 28;
    int maxThreads = 512;  // number of threads per block
    int whichKernel = 5;
    int maxBlocks = 64;
    int cpuFinalThreshold = 1;

    if (argc > 1) {
        int option = atoi(argv[1]);
        if((option >=0) && (option<6)) whichKernel = option;
        printf("Kernel option = %d\n", whichKernel);
        if(argc > 2){
            n = atoi(argv[2]);
            printf("n = %d\n", n);
        }
    }

    size_t bytes = n * sizeof(float);
    float *h_idata = (float *) malloc(bytes);
    for(int i = 0; i < n; i++)
        h_idata[i] = 1.0;

    int numBlocks = 0;
    int numThreads = 0;
    getNumBlocksAndThreads(whichKernel, n, maxBlocks, maxThreads, numBlocks, numThreads);

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(numBlocks*sizeof(float));

    printf("%d blocks\n\n", numBlocks);

    // allocate device memory and data
    float *d_idata = NULL;
    float *d_odata = NULL;

    checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
    checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(float)));

    // copy data directly to device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_odata, h_idata, numBlocks*sizeof(float), hipMemcpyHostToDevice));

    // warm-up
    reduce<float>(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));
    reduce<float>(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);
    bool needReadBack = true;
    float gpu_result = 0;
    // sum partial block sums on GPU
    int s=numBlocks;
    int kernel = whichKernel;

    while (s > cpuFinalThreshold)
    {
        int threads = 0, blocks = 0;
        getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);

        reduce<float>(s, threads, blocks, kernel, d_odata, d_odata);

        if (kernel < 3)
        {
            s = (s + threads - 1) / threads;
        }
        else
        {
            s = (s + (threads*2-1)) / (threads*2);
        }
    }

    if (s > 1)
    {
        // copy result from device to host
        checkCudaErrors(hipMemcpy(h_odata, d_odata, s * sizeof(float), hipMemcpyDeviceToHost));

        for (int i=0; i < s; i++)
        {
            gpu_result += h_odata[i];
        }

        needReadBack = false;
    }
    if (needReadBack)
    {
        // copy final sum from device to host
        checkCudaErrors(hipMemcpy(&gpu_result, d_odata, sizeof(float), hipMemcpyDeviceToHost));
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    printf("sum = %.1f\n", gpu_result);
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time: %f ms\n", milliseconds);

    free(h_idata);
    free(h_odata);

    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    return 0;
}
